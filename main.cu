#include "hip/hip_runtime.h"
﻿#include<glew/include/GL/glew.h>
#include<GLFW/glfw3.h>
#include<iostream>
#include<cmath>
#include"Header/error_check.h"
#include<chrono>
#include<thread>
#include"Header/mouse.h"
#include"Header/particleInjector.h"
#include"Header/parameters.h"

//#include""
//#include<hip/hip_runtime.h>

const char* vertexShader =
"#version 330 core\n"
"layout(location = 0) in vec4 position;\n"
"void main(){\n"
"gl_Position = position;\n"
"}\0\n";

const char* fragmentShader =
"#version 330 core\n"
"out vec4 color;\n"
"void main(){\n"
"color = vec4(1.0f , 0.5f , 0.2f , 1.0f);\n"
"}\0\n";


bool release{ false };

	void updateVBPositions(glm::vec3 dist, unsigned int pos, float* vertexbuffer)
	{
		unsigned int vb_pos = (NOOFTRIANGLES + 1) * 2 * pos;

		for (; vb_pos < (NOOFTRIANGLES + 1) * 2 * (pos + 1); vb_pos += 2)
		{
			vertexbuffer[vb_pos] = vertexbuffer[vb_pos] + dist.x;
			vertexbuffer[vb_pos + 1] = vertexbuffer[vb_pos + 1] + dist.y;
			//printf("position: %i and position in vertex buffer: %i\n", pos, vb_pos);
		}
	}

	void calcPosition(particle* particles, float* vertexbuffer,unsigned int count)
	{
		//unsigned int pos = blockDim.x * blockIdx.x + threadIdx.x;
        for (int pos = 0; pos < count; pos++)
        {
            if (particles[pos].injected)
            {
                float vX{ particles[pos].m_velocity.x }, vY(particles[pos].m_velocity.y);
                glm::vec3 dist = glm::vec3(vX * particles[pos].timestep + 0.5f * particles[pos].m_accelaration.x * particles[pos].timestep * particles[pos].timestep,
                    vY * particles[pos].timestep + 0.5f * particles[pos].m_accelaration.y * particles[pos].timestep * particles[pos].timestep, 0);

                if ((particles[pos].m_velocity.x == 0 || particles[pos].m_velocity.y == 0) && particles[pos].collision.occur)
                {
                    if (particles[pos].m_velocity.x == 0)
                    {
                        dist.x = 0;
                    }
                    if (particles[pos].m_velocity.y == 0)
                    {
                        dist.y = 0;
                    }
                }
                particles[pos].m_position = particles[pos].m_position + dist;

                updateVBPositions(dist, pos, vertexbuffer);
            }
        }
	}

	void calcVelocity(particle* particles, unsigned int count)
	{
		//unsigned int pos = blockDim.x * blockIdx.x + threadIdx.x;
        for (int pos = 0; pos < count; pos++)
        {
            if (particles[pos].injected)
            {
                float aX{ particles[pos].m_accelaration.x }, aY(particles[pos].m_accelaration.y);
                glm::vec3 velocity = glm::vec3(aX * particles[pos].timestep, aY * particles[pos].timestep, 0);

                if (!particles[pos].collision.occur)
                {
                    particles[pos].m_velocity = particles[pos].m_velocity + velocity;
                }
                else
                {
                    //particles[pos].m_velocity = particles[pos].collision.m_velocity + velocity; //maybe try?
                    particles[pos].m_velocity = particles[pos].collision.m_velocity;
                }
            }
        }
	}

	void calcAccelaration(particle* particles, unsigned int count)
	{
		//unsigned int pos = blockDim.x * blockIdx.x + threadIdx.x;
        for (int pos = 0; pos < count; pos++)
        {
            if (particles[pos].injected)
            {
                //for now just gravity
                if (particles[pos].m_position.y < -0.8f)
                {
                    particles[pos].m_accelaration = glm::vec3(0.0f, 0.0f, 0.0f);
                }
                else
                {
                    particles[pos].m_accelaration = glm::vec3(0.0f, -0.5f, 0.0f);
                }
            }
        }


	}

	void collisionCheck(particle* particles, unsigned int count)
	{
		//unsigned int pos = blockDim.x * blockIdx.x + threadIdx.x;
        for (int pos = 0; pos < count; pos++)
        {
            if (particles[pos].injected)
            {
                bool WallCollisiony = particles[pos].m_position.y <= -1.0f || particles[pos].m_position.y >= 1.0f;
                bool WallCollisionx = particles[pos].m_position.x <= -1.0f || particles[pos].m_position.x >= 1.0f;
                if ((WallCollisionx || WallCollisiony) && !particles[pos].collision.occur)
                {
                    if (WallCollisiony)
                    {
                        particles[pos].collision.occur = true;
                        particles[pos].collision.m_velocity.y = -particles[pos].m_velocity.y;
                        particles[pos].collision.m_velocity.x = particles[pos].m_velocity.x;
                    }
                    if (WallCollisionx)
                    {
                        particles[pos].collision.occur = true;
                        particles[pos].collision.m_velocity.x = -particles[pos].m_velocity.x;
                        particles[pos].collision.m_velocity.y = particles[pos].m_velocity.y;
                    }
                }
                else
                {
                    particles[pos].collision.occur = false;
                }

            }
        }


	}


int main(void)
{
    GLFWwindow* window;

    /* Initialize the library */
    if (!glfwInit())
        return -1;


    /* Create a windowed mode window and its OpenGL context */
    window = glfwCreateWindow(1920, 1080, "Hello World", NULL, NULL);

    //setting callbacks
    glfwSetMouseButtonCallback(window, mouseButtonCallback);

    if (!window)
    {
        glfwTerminate();
        return -1;
    }

    /* Make the window's context current */
    glfwMakeContextCurrent(window);
    //glfwSwapInterval(1);

    if (glewInit() != GLEW_OK)
    {
        return 1;
    }

    /*
    std::cout << glGetString(GL_VERSION) << std::endl;

    float positions[2 * (NOOFTRIANGLES + 1)]{ -0.5,-0.5 };

    //getting the points of a circle
    int k = 2;
    std::cout << "(" << positions[0] << "," << positions[1] << ")" << std::endl;
    for (int angle = 0; angle < 360; angle += 360 / NOOFTRIANGLES)
    {
        positions[k] = (CIRCLESIZE * std::cos((PI / (float)180) * angle) / (float)960) - 0.5;
        positions[k + 1] = (CIRCLESIZE * std::sin((PI / (float)180) * angle) / (float)540) - 0.5;
        std::cout << "(" << positions[k] << "," << positions[k + 1] << ")" << std::endl;
        k += 2;

    }

    unsigned int indices[3 * NOOFTRIANGLES]{ 0 };
    unsigned int temp = 1;
    for (int i = 0; i < 3 * NOOFTRIANGLES - 1; i++)
    {
        if (i % 3 == 0)
            indices[i] = 0;
        else if (i % 3 == 1)
        {
            indices[i] = temp;
            temp++;
        }
        else
        {
            indices[i] = temp;
        }
        std::cout << indices[i] << std::endl;
    }
    indices[3 * NOOFTRIANGLES - 1] = 1;

    int success;
    char info[512];

    unsigned int vertex_shader;
    vertex_shader = glCreateShader(GL_VERTEX_SHADER);
    glShaderSource(vertex_shader, 1, &vertexShader, NULL);
    glCompileShader(vertex_shader);

    glGetShaderiv(vertex_shader, GL_COMPILE_STATUS, &success);
    if (!success)
    {
        glGetShaderInfoLog(vertex_shader, 512, NULL, info);
        std::cout << "compilation failed :: vertex shader " << std::endl << info << std::endl;
    }

    unsigned int fragment_shader;
    fragment_shader = glCreateShader(GL_FRAGMENT_SHADER);
    glShaderSource(fragment_shader, 1, &fragmentShader, NULL);
    glCompileShader(fragment_shader);

    glGetShaderiv(fragment_shader, GL_COMPILE_STATUS, &success);
    if (!success)
    {
        glGetShaderInfoLog(fragment_shader, 512, NULL, info);
        std::cout << "compilation failed :: fragment shader " << std::endl << info << std::endl;
    }

    unsigned int shaderProgram;
    shaderProgram = glCreateProgram();
    glAttachShader(shaderProgram, vertex_shader);
    glAttachShader(shaderProgram, fragment_shader);
    glLinkProgram(shaderProgram);

    glGetProgramiv(shaderProgram, GL_LINK_STATUS, &success);
    if (!success)
    {
        glGetProgramInfoLog(shaderProgram, 512, NULL, info);
        std::cout << "linking failed " << std::endl << info << std::endl;
    }

    unsigned int vertexarrayID;
    glGenVertexArrays(1, &vertexarrayID);
    glBindVertexArray(vertexarrayID);

    unsigned int vertexBufferID;
    glGenBuffers(1, &vertexBufferID);
    glBindBuffer(GL_ARRAY_BUFFER, vertexBufferID);
    glBufferData(GL_ARRAY_BUFFER, 2 * (NOOFTRIANGLES + 1) * sizeof(float), positions, GL_DYNAMIC_DRAW);

    unsigned int indexBufferID;
    GLCALL(glGenBuffers(1, &indexBufferID));
    GLCALL(glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, indexBufferID));
    GLCALL(glBufferData(GL_ELEMENT_ARRAY_BUFFER, 3 * NOOFTRIANGLES * sizeof(unsigned int), indices, GL_STATIC_DRAW));


    glEnableVertexAttribArray(0);
    glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, 2 * sizeof(float), (void*)0);

    glUseProgram(shaderProgram);

    */

    //initializing the particle injector
    particleInjector particleinjector;
    particleinjector.init();

    unsigned int count;
    //particle* d_particles;
    //float* d_vertexbufferdata;
    particle* particles(particleinjector.getParticlePointer());
    float* vertexbufferdata(particleinjector.getVertexBufferData());

    //hipMalloc(&d_particles,MAX_PARTICLES*sizeof(particle));
    //hipMalloc(&d_vertexbufferdata, MAX_PARTICLES* ((NOOFTRIANGLES + 1) * 2)*sizeof(float));
    //hipMemcpy(d_particles,particles, MAX_PARTICLES * sizeof(particle),hipMemcpyHostToDevice);
    //hipMemcpy(d_vertexbufferdata, vertexbufferdata, MAX_PARTICLES* ((NOOFTRIANGLES + 1) * 2) * sizeof(float), hipMemcpyHostToDevice);

    glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);

    while (!glfwWindowShouldClose(window))
    {
        glClear(GL_COLOR_BUFFER_BIT);

        handleMouseInputEvent(window,particleinjector);

        //calculations using cuda
        count = particleinjector.getParticleCount();
        unsigned int threads = (count >= 32 )? 32 : count;
        unsigned int blocks = (unsigned int)ceil((float)count/32.0f);



        //hipMemcpy(d_particles, particles, MAX_PARTICLES * sizeof(particle), hipMemcpyHostToDevice);
        //hipMemcpy(d_vertexbufferdata, vertexbufferdata, MAX_PARTICLES * ((NOOFTRIANGLES + 1) * 2) * sizeof(float), hipMemcpyHostToDevice);
        //cuda::calcAccelaration<<<blocks, threads>>>(d_particles);
        //cuda::calcVelocity<<<blocks, threads>>>(d_particles);
        //cuda::calcPosition<<<blocks, threads>>>(d_particles, d_vertexbufferdata);
        //hipDeviceSynchronize();
        //cuda::collisionCheck<<<blocks, threads>>>(d_particles);
        //hipMemcpy(particles, d_particles, MAX_PARTICLES * sizeof(particle), hipMemcpyDeviceToHost);
        //hipMemcpy(vertexbufferdata, d_vertexbufferdata, MAX_PARTICLES * ((NOOFTRIANGLES + 1) * 2) * sizeof(float), hipMemcpyDeviceToHost);

        calcAccelaration(particles,count);
        calcVelocity(particles, count);
        calcPosition(particles, vertexbufferdata, count);
        collisionCheck(particles, count);


        for (int h = 0; h < 10; h+=2)
            std::cout << vertexbufferdata[h] <<","<< vertexbufferdata[h+1] << std::endl;

        //rendering
        particleinjector.renderParticles();

        /*
        for (int t = 0; t < 2 * (NOOFTRIANGLES + 1); t+=2)
        {
            positions[t] += 0.001f;

            positions[t + 1] += 0.001f;
        }
        glBufferData(GL_ARRAY_BUFFER, 2 * (NOOFTRIANGLES + 1) * sizeof(float), positions, GL_DYNAMIC_DRAW);

        //glDrawArrays(GL_TRIANGLES, 0, (NOOFTRIANGLES + 1));
        GLCALL(glDrawElements(GL_TRIANGLES, NOOFTRIANGLES * 3, GL_UNSIGNED_INT, 0));
        */

        /* Swap front and back buffers */
        glfwSwapBuffers(window);

        /* Poll for and process events */
        glfwPollEvents();
    }

    /*
    glDeleteShader(vertex_shader);
    glDeleteShader(fragment_shader);

    GLCALL(glDeleteBuffers(1, &vertexBufferID));
    GLCALL(glDeleteBuffers(1, &indexBufferID));
    GLCALL(glDeleteVertexArrays(1, &vertexarrayID));
    */

    //hipFree(d_particles);
    //hipFree(d_vertexbufferdata);

    glfwTerminate();
    return 0;
}

